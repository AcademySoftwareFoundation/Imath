#include "hip/hip_runtime.h"
//
// SPDX-License-Identifier: BSD-3-Clause
// Copyright Contributors to the OpenEXR Project.
//

#ifdef NDEBUG
#    undef NDEBUG
#endif

#include "testVecCUDA.h"
#include <ImathFun.h>
#include <ImathVec.h>
#include <cuda/std/cassert>
#include <cuda/std/cmath>

// Include ImathForward *after* other headers to validate forward declarations
#include <ImathForward.h>

using namespace cuda::std;
using namespace IMATH_INTERNAL_NAMESPACE;

namespace
{

template <class T>
void
testLength2T ()
{
    const T s = sqrt (numeric_limits<T>::min ());
    const T e = 4 * numeric_limits<T>::epsilon ();

    Vec2<T> v;

    v = Vec2<T> (0, 0);
    assert (v.length () == 0);
    assert (v.normalized ().length () == 0);

    v = Vec2<T> (3, 4);
    assert (v.length () == 5);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    v = Vec2<T> (3000, 4000);
    assert (v.length () == 5000);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    T t = s * (1 << 4);

    v = Vec2<T> (t, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec2<T> (0, t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec2<T> (-t, -t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (
        v.length (), t * sqrt (2), t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    t = s / (1 << 4);

    v = Vec2<T> (t, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec2<T> (0, t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec2<T> (-t, -t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (
        v.length (), t * sqrt (2), t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    t = s / (1 << 20);

    v = Vec2<T> (t, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec2<T> (0, t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec2<T> (-t, -t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (
        v.length (), t * sqrt (2), t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
}

template <class T>
void
testLength3T ()
{
    const T s = sqrt (numeric_limits<T>::min ());
    const T e = 4 * numeric_limits<T>::epsilon ();

    Vec3<T> v;

    v = Vec3<T> (0, 0, 0);
    assert (v.length () == 0);
    assert (v.normalized ().length () == 0);

    v = Vec3<T> (3, 4, 0);
    assert (v.length () == 5);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    v = Vec3<T> (3000, 4000, 0);
    assert (v.length () == 5000);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    v = Vec3<T> (1, -1, 1);
    assert (
        IMATH_INTERNAL_NAMESPACE::equal (v.length (), 1 * sqrt (3), e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    v = Vec3<T> (1000, -1000, 1000);
    assert (IMATH_INTERNAL_NAMESPACE::equal (
        v.length (), 1000 * sqrt (3), 1000 * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    T t = s * (1 << 4);

    v = Vec3<T> (t, 0, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec3<T> (0, t, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec3<T> (0, 0, t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec3<T> (-t, -t, -t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (
        v.length (), t * sqrt (3), t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    t = s / (1 << 4);

    v = Vec3<T> (t, 0, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec3<T> (0, t, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec3<T> (0, 0, t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec3<T> (-t, -t, -t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (
        v.length (), t * sqrt (3), t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    t = s / (1 << 20);

    v = Vec3<T> (t, 0, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec3<T> (0, t, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec3<T> (0, 0, t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec3<T> (-t, -t, -t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (
        v.length (), t * sqrt (3), t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
}

template <class T>
void
testLength4T ()
{
    const T s = sqrt (numeric_limits<T>::min ());
    const T e = 4 * numeric_limits<T>::epsilon ();

    Vec4<T> v;

    v = Vec4<T> (0, 0, 0, 0);
    assert (v.length () == 0);
    assert (v.normalized ().length () == 0);

    v = Vec4<T> (3, 4, 0, 0);
    assert (v.length () == 5);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    v = Vec4<T> (3000, 4000, 0, 0);
    assert (v.length () == 5000);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    v = Vec4<T> (1, -1, 1, 1);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), 2, e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    v = Vec4<T> (1000, -1000, 1000, 1000);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), 2000, 1000 * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    T t = s * (1 << 4);

    v = Vec4<T> (t, 0, 0, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (0, t, 0, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (0, 0, t, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (0, 0, 0, t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (-t, -t, -t, -t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t * 2, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    t = s / (1 << 4);

    v = Vec4<T> (t, 0, 0, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (0, t, 0, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (0, 0, t, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (0, 0, 0, t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (-t, -t, -t, -t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t * 2, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));

    t = s / (1 << 20);

    v = Vec4<T> (t, 0, 0, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (0, t, 0, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (0, 0, t, 0);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (0, 0, 0, t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
    v = Vec4<T> (-t, -t, -t, -t);
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.length (), t * 2, t * e));
    assert (IMATH_INTERNAL_NAMESPACE::equal (v.normalized ().length (), 1, e));
}

} // namespace

void
testVecCUDA ()
{
    testLength2T<float> ();
    testLength2T<double> ();
    testLength3T<float> ();
    testLength3T<double> ();
    testLength4T<float> ();
    testLength4T<double> ();

    // Test template parameter checks pass an alias of a correct type.
    using R = double;
    testLength4T<R> ();
}
